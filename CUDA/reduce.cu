
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void evenReduce(int *a, int *b, int numP)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < numP)
		b[index] = a[index * 2] + a[index * 2 + 1];
}

__global__ void oddReduce(int *a, int *b, int numP)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < numP) 
	{
	if(index != numP - 1)
			{ 
				b[index] = a[index * 2] + a[index * 2 + 1];
			}
	else 
			{
				/*puts the remaining value that doesn't have a pair in the right index*/
				b[index] = a[index * 2];
			}
	}		
} 

#define LENGTH 512
#define BLOCK_THREADS 512

int main()
{
	double length = LENGTH;
	int numP, l;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
   	
   	/*create arrays for host and GPU*/
   	int *a, *b, *k_b, *k_a;
	int size = length * sizeof( int );

	a = (int *)malloc( size );
	b = (int *)malloc( size );

	hipMalloc( (void **) &k_a, size );
	hipMalloc( (void **) &k_b, size );

   	/*initialize the array*/
	for( int i = 0; i < length; i++ )
	{
		a[i] = i;
		b[i] = 0;
	}

	/*array debug*/
	// printf("A:\n");
	// for(int i=0; i< length; i++)
	// {
	// 	printf("%d ", a[i]);
	// }

	/* copy inputs to device */
	hipMemcpy(k_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy(k_b, b, size, hipMemcpyHostToDevice );

 	dim3 dimGrid( 1, 1 );
 	dim3 dimBlock(BLOCK_THREADS, 1);
	
	/*Since each thread does 2 additions there are log2(N) iterations.*/
	int gates = ceil(log(length) / log(2));
	
	hipEventRecord(start);
	
	for(int i=0; i < gates; i++) {

		/*get the number of threads needed. Ceiling used for odd array lengths*/
		numP = ceil(length/2);
				
		l = (int)length;
		/*when threads are divisble by 2 use less code...*/
		if( l % 2 == 0)
			evenReduce<<<dimGrid,dimBlock>>>(k_a, k_b, numP);
		else 
			oddReduce<<<dimGrid,dimBlock>>>(k_a, k_b, numP);
		
		/*last # of threads will equal next array length to compute*/
		length = numP;
		
		// printf("\niteration %d reduction is\n", i + 1);
		// cudaMemcpy(b, k_b, size, cudaMemcpyDeviceToHost );
		// for(int i=0; i< N; i++)
		// 	{
		// 	printf("%d ", b[i]);
		// 	}

		/*send array b's data back to a after each iteration*/
		hipMemcpy(k_a, k_b, size, hipMemcpyDeviceToDevice );
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	/* copy result back to host*/

	hipMemcpy(b, k_b, size, hipMemcpyDeviceToHost );
	printf("\nFinal reduction is %d\n", b[0]);
	printf("\nThis took %f milliseconds\n", milliseconds);

	/* clean up */

	free(a);
	free(b);
	hipFree( k_a );
	hipFree( k_b );
	
	return 0;
} 

