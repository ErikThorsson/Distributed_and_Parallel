
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = threadIdx.x;
	c[index] = a[index] + b[index];
}

#define dim 3

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	//since we will be sending an array to represend the matrix.
	int size = dim * dim * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < dim * dim; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	printf("A and B are:\n");
	for(int i=0; i< dim * dim; i++)
	{
		if(i%dim == 0)
			printf("\n");
		printf("%d ", a[i]);
	}

	/* copy inputs to device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	dim3 dimBlock(10, 1 );
 	dim3 dimGrid( 1, 1 );
	
	vector_add<<<dimGrid,dimBlock>>>( d_a, d_b, d_c);

	/* copy result back to host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	printf("\n\nTheir sum =\n");
	for(int i=0; i< dim * dim; i++)
	{
		if(i%dim == 0)
			printf("\n");
		printf("%d ", c[i]);
	}

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} 

